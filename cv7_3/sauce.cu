#include "hip/hip_runtime.h"
#include "sauce.hpp"


Compute::Compute(int wordLen) {
    this->wordLen= wordLen;
    this->numOfWords= calculateNumberOfWords(wordLen);
}

unsigned long long Compute::calculateNumberOfWords(int numberOfLetters) {
    return (unsigned long long)pow(LETTERS_SIZE, numberOfLetters);

}

void Compute::gpuHandler(unsigned long long int numOfWOrd) {
    char *host_word, host_letter, *cuda_word, *cuda_letter;
    host_letter= &LETTERS[0];
//    strcpy(LETTERS)


//    hipError_t cerr;
//    int threads = 256;
//    int blocks = ( this->wordLen + threads - 1 ) / threads;
//
//    // Memory allocation in GPU device
//    char *lett;
//    cerr = hipMalloc( &lett, LETTERS->size()*sizeof(char) );
//    if ( cerr != hipSuccess )
//        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );
//
//    // Copy data from PC to GPU device
//    cerr = hipMemcpy( lett, LETTERS, LETTERS->size()* sizeof( char ), hipMemcpyHostToDevice );
//    if ( cerr != hipSuccess )
//        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );
//
//    // Grid creation
//    wordCompute<<< blocks, threads >>>( lett );
//
//    if ( ( cerr = hipGetLastError() ) != hipSuccess )
//        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );
//
//    // Copy data from GPU device to PC
//    cerr = hipMemcpy( P, cudaP, this->wordLen * sizeof( float ), hipMemcpyDeviceToHost );
//    if ( cerr != hipSuccess )
//        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );
//
//    // Free memory
//    hipFree( cudaP );
}

__global__ void wordCompute( char* lett, const int gridIdx, const int gridDim) {
    dim3 cudaBlockSize(256);


}
