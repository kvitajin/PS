#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cmath>
#include <chrono>
#include <zconf.h>
#include <wait.h>
#include <fstream>
#include <vector>
#include <thread>
#include "md5.hpp"
#include "sauce.hpp"


//
//
//
//void forkController(int wordLen, int firstWord, int numOfWord, bool makeHash, bool print){
////    for (int i = 0; i < numOfWord; ++i) {
////        generateString(firstWord+i, wordLen, makeHash, print);
////    }
//}
//


//
//void gridHeader(int wordLen, int numberOfForks, bool makeHash, bool print){             //TODO edit it
//    std::vector<std::thread> threads;
//    int numberOfWords = calculateNumberOfWords(wordLen);
//    int partSize = floor(numberOfWords/numberOfForks);
//    for (int i = 0; i < numberOfForks; ++i) {
//        if (i+1==numberOfForks){
//            int lastPart = numberOfWords-i*partSize;
//            threads.emplace_back(std::thread(forkController,wordLen, partSize*i, lastPart, makeHash, print));
//        }
//        threads.emplace_back(std::thread(forkController,wordLen, partSize*i, partSize, makeHash, print));
//    }
//
//    for (std::thread & th : threads){
//        if (th.joinable())
//            th.join();
//    }
//
//}
//
//void makeCSV(int wordLen, int numberOfForks, bool makeHash, bool print){
//    std::chrono::steady_clock sc;
//    std::string fileName;
//    fileName.push_back((char)wordLen + 48);
//    fileName.append("znaky.csv");
//    std::ofstream output;
//    output.open(fileName);
//    output<<"THREADS\tTIME\n";
//    for (int j = 1; j < numberOfForks+1; ++j) {
//        auto start=sc.now();
//        gridHeader(wordLen, j, makeHash, print);
//        auto end=sc.now();
//        auto time_span = static_cast<std::chrono::duration<double>>(end - start);
//        output<<j <<"\t"<<time_span.count()<<std::endl;
//    }
//    output.close();
//
//}

int main() {
    bool print = false;
    bool makeHash = true;
    int wordLen=4;
    int numberOfForks= 100;
//    makeCSV( wordLen, numberOfForks, makeHash, print);
    std::cout<<md5("aa");
    Compute compute(wordLen);
    return 0;
}
